#include "hip/hip_runtime.h"
#include    <wb.h>

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
			       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
	int ty=blockIdx.y*blockDim.y+threadIdx.y;
	int tx=blockIdx.x*blockDim.x+threadIdx.x;
	if (ty<numCRows && tx<numCColumns) {
        float sum=0.0;
        int despA,despAEnd,despB;
		despA=ty*numAColumns; despB=tx;
        despAEnd=despA+numAColumns;
        for(;despA<despAEnd;despA++,despB+=numBColumns) {
			sum+=A[despA]*B[despB];
		}
		C[ty*numCColumns+tx]=(float)sum;
	}

}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)
    int sizeA,sizeB,sizeC;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    sizeA=numARows*numAColumns*sizeof(float);
    sizeB=numBRows*numBColumns*sizeof(float);
    sizeC=numCRows*numCColumns*sizeof(float);
    //@@ Allocate the hostC matrix
    hostC=(float *)malloc(sizeC);

    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
    wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc((void**)&deviceA,sizeA));
    wbCheck(hipMalloc((void**)&deviceB,sizeB));
    wbCheck(hipMalloc((void**)&deviceC,sizeC));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy(deviceA,hostA,sizeA,hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceB,hostB,sizeB,hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
#define TX 32
#define TY 16
    dim3 DimGrid((numCColumns-1)/TX + 1, (numCRows-1)/TY+1, 1);
    dim3 DimBlock(TX, TY, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiply<<<DimGrid,DimBlock>>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBRows,numBColumns,numCRows,numCColumns);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy(hostC,deviceC,sizeC,hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree(deviceA));wbCheck(hipFree(deviceB));wbCheck(hipFree(deviceC));

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

