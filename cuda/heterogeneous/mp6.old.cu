#include "hip/hip_runtime.h"
#include    <wb.h>

// Check ec2-174-129-21-232.compute-1.amazonaws.com:8080/mp/6 for more information

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

#define MASK_WIDTH  5
#define Mask_radius MASK_WIDTH/2
#define CHANNELS 3
#define TILE_SIZE 5

//@@ INSERT CODE HERE
//def clamp(x, start, end)
//      return min(max(x, start), end)
//    end
//#define clamp(x,s,e) ((x>s?x:s)<e?(x>s?x:s):e)
#define MIN(x,y) (x<y?x:y)
#define MAX(x,y) (x>y?x:y)
#define clamp(x,s,e) MIN(MAX(x,s),e)
#define _N(y,x,k) N[(y)*totalWidth+(x)*CHANNELS+k]
#define _P(y,x,k) P[(y)*totalWidth+(x)*CHANNELS+k]

//__device__ float clamp(float x, float s, float e) {
//	float max=(x>s?x:s);
//	return (max<e?max:e);
//}
//__global__ void convolution2DImage(float *N, const float __restrict__ *M, float *P, int Width, int Height) {
__global__ void convolution2DImage(float *N, const float *M, float *P, int Width, int Height) {
	__shared__ float N_ds[TILE_SIZE+MASK_WIDTH-1][TILE_SIZE+MASK_WIDTH-1][CHANNELS];
	int tx=threadIdx.x, ty=threadIdx.y, bx=blockIdx.x, by=blockIdx.y, totalWidth=CHANNELS*Width;

	int indx=bx*blockDim.x+tx,
		indy=by*blockDim.y+ty;
	//++++++++++ QUITAR
	int f=2,c=2;
	if (indy==f && indx==c) {
		printf("indy: %d, indx: %d\n",indy,indx);
		for(int i=f-Mask_radius;i<=f+2*Mask_radius;i++) {
			for(int j=c-Mask_radius;j<=c+2*Mask_radius;j++) {
				//printf("Indice : %d",(i)*totalWidth+(j)*CHANNELS);
				printf("N[%d][%d]=%f.",i,j,N[(i)*totalWidth+(j)*CHANNELS]);
			}
			printf("\n");
		}
	}
	__syncthreads();
	//++++++++++ QUITAR
	//++++++++++ QUITAR
		__syncthreads();
		int f=0,c=192;
		if (indy==f && indx==c) {
			printf("indy: %d, indx: %d\n",indy,indx);
			for(int i=0;i<20;i++) {
				for(int j=190;j<200;j++) {
					//printf("Indice : %d",(i)*totalWidth+(j)*CHANNELS);
					printf("%f;",_N(i,j,0));
				}
				printf("\n");
			}
		}
		__syncthreads();
	//++++++++++ QUITAR
	if (indx<Width && indy<Height) {
		int haloIndX,haloIndY;
		if (tx>=TILE_SIZE-Mask_radius) { // left
			haloIndX=tx-(TILE_SIZE-Mask_radius);
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+Mask_radius][haloIndX][k]=(indx-TILE_SIZE<0 ? 0 : _N(indy,indx-TILE_SIZE,k));
			}
		}
		if (ty>=TILE_SIZE-Mask_radius) { // up
			haloIndY=ty-(TILE_SIZE-Mask_radius);
			for(int k=0;k<CHANNELS;k++) {
				N_ds[haloIndY][tx+Mask_radius][k]=(indy-TILE_SIZE<0 ? 0 : _N(indy-TILE_SIZE,indx,k));
			}
		}
		if (tx<Mask_radius) { // right
			haloIndX=Mask_radius+TILE_SIZE+tx;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+Mask_radius][haloIndX][k]=(indx+TILE_SIZE>=Width ? 0 : _N(indy,indx+TILE_SIZE,k));
			}
		}
		if (ty<Mask_radius) { // bottom
			haloIndY=Mask_radius+TILE_SIZE+ty;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[haloIndY][tx+Mask_radius][k]=(indy+TILE_SIZE>=Height ? 0 : _N(indy+TILE_SIZE,indx,k));
			}
		}
		if (tx<Mask_radius && ty<Mask_radius) { // upper left corner
			for(int k=0;k<CHANNELS;k++) {
				if (indy-Mask_radius<0 || indx-Mask_radius<0) {
					N_ds[ty][tx][k]=0;
				} else {
					N_ds[ty][tx][k]=_N(indy-Mask_radius,indx-Mask_radius,k);
				}
			}
		}
		if (tx<Mask_radius && ty>=TILE_SIZE-Mask_radius) { // bottom left corner
			for(int k=0;k<CHANNELS;k++) {
				if (indy+Mask_radius>=Height || indx-Mask_radius<0) {
					N_ds[Mask_radius+ty+Mask_radius][tx][k]=0;
				} else {
					N_ds[Mask_radius+ty+Mask_radius][tx][k]=_N(indy+Mask_radius,indx-Mask_radius,k);
				}
			}
		}
		if (tx>=TILE_SIZE-Mask_radius && ty>=TILE_SIZE-Mask_radius) { //bottom right corner
			for(int k=0;k<CHANNELS;k++) {
				if (indy+Mask_radius>=Height || indx+Mask_radius>=Width) {
					N_ds[Mask_radius+ty+Mask_radius][Mask_radius+tx+Mask_radius][k]=0;
				} else {
					N_ds[Mask_radius+ty+Mask_radius][Mask_radius+tx+Mask_radius][k]=_N(indy+Mask_radius,indx+Mask_radius,k);
				}
			}
		}
		if (tx>=TILE_SIZE-Mask_radius && ty<Mask_radius) { // upper right corner
			for(int k=0;k<CHANNELS;k++) {
				if (indy-Mask_radius<0 || indx+Mask_radius>=Width) {
					N_ds[ty][Mask_radius+tx+Mask_radius][k]=0;
				} else {
					N_ds[ty][Mask_radius+tx+Mask_radius][k]=_N(indy-Mask_radius,indx+Mask_radius,k);
				}
			}
		}
		for(int k=0;k<CHANNELS;k++) {
			N_ds[ty+Mask_radius][tx+Mask_radius][k]=_N(indy,indx,k);
		}
		__syncthreads();
		//++++++++++++++++++ QUITAR
		if (indy==f && indx==c) {
			printf("\n");
			printf("indy: %d, indx: %d\n",indy,indx);
			printf("Rangos i: %d--%d, j: %d-%d.\n",f-Mask_radius,f+2*Mask_radius+TILE_SIZE/2,c-Mask_radius,c+2*Mask_radius+TILE_SIZE/2);
			for(int i=f-Mask_radius;i<=f+2*Mask_radius+TILE_SIZE/2;i++) {
				for(int j=c-Mask_radius;j<=c+2*Mask_radius+TILE_SIZE/2;j++) {
					//printf("Indice : %d",(i)*totalWidth+(j)*CHANNELS);
					printf("N_ds[%d][%d]=%f.",i,j,N_ds[i][j][0]);
				}
				printf("\n");
			}
		}
		//++++++++++++++++++ QUITAR
		//++++++++++++++++++ QUITAR
					if (indy==f && indx==c) {
						printf("\n");
						printf("indy: %d, indx: %d\n",indy,indx);
						printf("Rangos i: %d--%d, j: %d-%d.\n",f-Mask_radius,f+2*Mask_radius+TILE_SIZE/2,c-Mask_radius,c+2*Mask_radius+TILE_SIZE/2);
						for(int i=0;i<20;i++) {
							for(int j=0;j<20;j++) {
								//printf("Indice : %d",(i)*totalWidth+(j)*CHANNELS);
								printf("%f;",N_ds[i][j][0]);
							}
							printf("\n");
						}
					}
					__syncthreads();
			//++++++++++++++++++ QUITAR
		float PValue[CHANNELS];
		for(int k=0;k<CHANNELS;k++) {
			PValue[k]=0;
			for(int i=0;i<MASK_WIDTH;i++) {
				for(int j=0;j<MASK_WIDTH;j++) {
					PValue[k]+=N_ds[ty+i][tx+j][k]*M[i*MASK_WIDTH+j];
				}
			}
			//P[indy*totalWidth+indx*CHANNELS+k]=clamp(PValue[k],0,1);
			_P(indy,indx,k)=clamp(PValue[k],0,1);
		}
	}
}


int main(int argc, char* argv[]) {
    wbArg_t arg;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    arg = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(arg, 0);
    inputMaskFile = wbArg_getInputFile(arg, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");


    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");

    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
    dim3 DimGrid((imageWidth-1)/TILE_SIZE + 1, ( imageHeight-1)/TILE_SIZE+1, 1);
    dim3 DimBlock(TILE_SIZE, TILE_SIZE, 1);
    printf("lanzando el kernel: \n");
    printf("tam img: %d,%d, channels: %d\n",imageWidth,imageHeight,imageChannels);
    convolution2DImage<<<DimGrid,DimBlock>>>(deviceInputImageData,deviceMaskData,deviceOutputImageData,imageWidth,imageHeight);
    wbTime_stop(Compute, "Doing the computation on the GPU");


    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(arg, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}
