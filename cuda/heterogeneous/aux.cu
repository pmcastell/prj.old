#include "hip/hip_runtime.h"
__global__ void _convolution2DImage(float *N, const float *M, float *P, int Width, int Height, int Mask_Width) {
	__shared__ float N_ds[TILE_SIZE+Mask_width-1][TILE_SIZE+Mask_width-1][CHANNELS];
	int tx=threadIdx.x, ty=threadIdx.y, bx=blockIdx.x, by=blockIdx.y, totalWidth=CHANNELS*Width, totalTileSize=TILE_SIZE*CHANNELS;
	//int j=tx*blockIdx.x*CHANNELS+
	int indx=(bx*blockDim.x+tx)*CHANNELS;
	int indy=by*blockDim.y+ty;
	if (indx<totalWidth && indy<Height) {
		int halo_indexX,
			//n=CHANNELS*Mask_radius,
			halo_indexY;
		if (tx>=TILE_SIZE-Mask_radius) { // left
			halo_indexX=indx-totalTileSize;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx-(TILE_SIZE-Mask_radius)][k]=(halo_indexX<0 ? 0 :N[indy*totalWidth+halo_indexX+k]);
			}
		}
		if (ty>=TILE_SIZE-Mask_radius) { // up
			halo_indexY=indy-TILE_SIZE*totalWidth;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty-(TILE_SIZE-Mask_radius)][tx][k]=(halo_indexY<0 ? 0 :N[halo_indexY*totalWidth+indx+k]);
			}
		}
		if (tx<Mask_radius) { // right
			halo_indexX=indx+totalTileSize;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx+TILE_SIZE][k]=(halo_indexX>=totalWidth ? 0:N[indy*totalWidth+halo_indexX+k]);
			}
		}
		if (ty<Mask_radius) {  //bottom
			halo_indexY=indy+TILE_SIZE*totalWidth;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+(TILE_SIZE+Mask_radius)][tx][k]=(halo_indexY>=TILE_SIZE?0:N[halo_indexY*totalWidth+indx+k]);
			}
		}
		if (tx<Mask_radius && ty<Mask_radius) { //left upper corner
			int ind=(indy-Mask_radius)*totalWidth+indx-Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx][k]=(ind<0 ? 0: N[ind+k]);
			}
		}
		if (tx<Mask_radius && ty>=TILE_SIZE-Mask_radius) { //left bottom corner
			int ind=(indy+Mask_radius)*totalWidth+indx-Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+Mask_radius][tx][k]=(ind>=totalWidth? 0: N[ind+k]);
			}
		}
		if (tx>=TILE_SIZE-Mask_radius && ty<Mask_radius) { // right upper corner
			int ind=(indy-Mask_radius)*totalWidth+indx+Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx+Mask_radius][k]=(ind>=totalWidth? 0: N[ind+k]);
			}
		}
		if (tx>=TILE_SIZE-Mask_radius && ty>=TILE_SIZE-Mask_radius) { //right bottom corner
			int ind=(indy+Mask_radius)*totalWidth+indx+Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+Mask_radius][tx+Mask_radius][k]=(ind>=totalWidth?0:N[ind+k]);
			}
		}
		for(int k=0;k<CHANNELS;k++) {
			N_ds[ty+Mask_radius][tx+Mask_radius][k]=N[indy*totalWidth+indx+k];
		}
		__syncthreads();
		float PValue[CHANNELS]={0,0,0};
		for(int k=0;k<CHANNELS;k++) {
			for(int i=0;i<Mask_width;i++) {
				for(int j=0;j<Mask_width;j++) {
					PValue[k]+=N_ds[ty+i][tx+j][k]*M[i*Mask_width+j];
				}
			}
			P[indy*totalWidth+indx+k]=clamp(PValue[k],0,1);
		}

	}
}


__global__ void convolution2DImage(float *N, const float *M, float *P, int Width, int Height, int Mask_Width) {
	__shared__ float N_ds[TILE_SIZE+Mask_width-1][TILE_SIZE+Mask_width-1][CHANNELS];
	int tx=threadIdx.x, ty=threadIdx.y, bx=blockIdx.x, by=blockIdx.y, totalWidth=CHANNELS*Width, totalTileSize=TILE_SIZE*CHANNELS;
	//int j=tx*blockIdx.x*CHANNELS+
	int indx=(bx*blockDim.x+tx)*CHANNELS;
	int indy=by*blockDim.y+ty;
	if (indx<totalWidth && indy<Height) {
		int halo_indexX,
			//n=CHANNELS*Mask_radius,
			halo_indexY;
		if (tx>=TILE_SIZE-Mask_radius) { // left
			halo_indexX=indx-totalTileSize;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx-(TILE_SIZE-Mask_radius)][k]=(halo_indexX<0 ? 0 :N[indy*totalWidth+halo_indexX+k]);
			}
		}
		if (ty>=TILE_SIZE-Mask_radius) { // up
			halo_indexY=indy-TILE_SIZE*totalWidth;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty-(TILE_SIZE-Mask_radius)][tx][k]=(halo_indexY<0 ? 0 :N[halo_indexY*totalWidth+indx+k]);
			}
		}
		if (tx<Mask_radius) { // right
			halo_indexX=indx+totalTileSize;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx+TILE_SIZE+Mask_radius][k]=(halo_indexX>=totalWidth ? 0:N[indy*totalWidth+halo_indexX+k]);
			}
		}
		if (ty<Mask_radius) {  //bottom
			halo_indexY=indy+TILE_SIZE*totalWidth;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+(TILE_SIZE+Mask_radius)][tx][k]=(halo_indexY>=TILE_SIZE?0:N[halo_indexY*totalWidth+indx+k]);
			}
		}
		if (tx<Mask_radius && ty<Mask_radius) { //left upper corner
			int ind=(indy-Mask_radius)*totalWidth+indx-Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx][k]=(ind<0 ? 0: N[ind+k]);
			}
		}
		if (tx<Mask_radius && ty>=TILE_SIZE-Mask_radius) { //left bottom corner
			int ind=(indy+Mask_radius)*totalWidth+indx-Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+Mask_radius][tx][k]=(ind>=totalWidth? 0: N[ind+k]);
			}
		}
		if (tx>=TILE_SIZE-Mask_radius && ty<Mask_radius) { // right upper corner
			int ind=(indy-Mask_radius)*totalWidth+indx+Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty][tx+Mask_radius][k]=(ind>=totalWidth? 0: N[ind+k]);
			}
		}
		if (tx>=TILE_SIZE-Mask_radius && ty>=TILE_SIZE-Mask_radius) { //right bottom corner
			int ind=(indy+Mask_radius)*totalWidth+indx+Mask_radius*CHANNELS;
			for(int k=0;k<CHANNELS;k++) {
				N_ds[ty+Mask_radius][tx+Mask_radius][k]=(ind>=totalWidth?0:N[ind+k]);
			}
		}
		for(int k=0;k<CHANNELS;k++) {
			N_ds[ty+Mask_radius][tx+Mask_radius][k]=N[indy*totalWidth+indx+k];
		}
		__syncthreads();
		float PValue[CHANNELS]={0,0,0};
		for(int k=0;k<CHANNELS;k++) {
			for(int i=0;i<Mask_width;i++) {
				for(int j=0;j<Mask_width;j++) {
					PValue[k]+=N_ds[ty+i][tx+j][k]*M[i*Mask_width+j];
				}
			}
			P[indy*totalWidth+indx+k]=clamp(PValue[k],0,1);
		}

	}
}
__global__ void __convolution2DImage(float *N, const float *M, float *P, int Width, int Height, int Mask_Width) {
	int indy=blockIdx.y*blockDim.y+threadIdx.y;
	int indx=blockIdx.x*blockDim.x*CHANNELS+threadIdx.x;
	int totalWidth=Width*CHANNELS;
	//for(int k=0;k<CHANNELS;k++) {
		P[indy*totalWidth+indx+0]=0.458824;
		P[indy*totalWidth+indx+1]=0.290196;
		P[indy*totalWidth+indx+2]=0.137255;
	//}
}
