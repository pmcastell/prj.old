#include "hip/hip_runtime.h"
// MP 3: Due Sunday, Dec 30, 2012 at 11:59 p.m. PST
#include    <wb.h>
#define TILE_WIDTH 32
#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)
// Compute C = A * B
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
			             int numARows, int numAColumns,
			             int numBRows, int numBColumns,
			             int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    //@@ You have to use shared memory for this MP
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;
	//int Width=numAColumns;
	for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
		if (Row<numARows && m*TILE_WIDTH+tx<numAColumns) {
			ds_A[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
		} else {
			ds_A[ty][tx]=0;
		}
		if (m*TILE_WIDTH+ty<numBRows && Col<numBColumns) {
			ds_B[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
		} else {
			ds_B[ty][tx]=0;
		}
		__syncthreads();
		if (Row<numCRows && Col<numCColumns) {
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Pvalue += ds_A[ty][k] * ds_B[k][tx];
			}
		}
		__syncthreads();
	}
	if (Row<numCRows && Col<numCColumns) {
		C[Row*numCColumns+Col] = Pvalue;
	}
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)
    int sizeA,sizeB,sizeC;
    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    sizeA=numARows*numAColumns*sizeof(float);
    sizeB=numBRows*numBColumns*sizeof(float);
    sizeC=numCRows*numCColumns*sizeof(float);
    //@@ Allocate the hostC matrix
    hostC=(float *)malloc(sizeC);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
    wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc((void**)&deviceA,sizeA));
    wbCheck(hipMalloc((void**)&deviceB,sizeB));
    wbCheck(hipMalloc((void**)&deviceC,sizeC));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy(deviceA,hostA,sizeA,hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceB,hostB,sizeB,hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here

    dim3 DimGrid((numCColumns-1)/TILE_WIDTH + 1, (numCRows-1)/TILE_WIDTH+1, 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<DimGrid,DimBlock>>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBRows,numBColumns,numCRows,numCColumns);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy(hostC,deviceC,sizeC,hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree(deviceA));wbCheck(hipFree(deviceB));wbCheck(hipFree(deviceC));
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

