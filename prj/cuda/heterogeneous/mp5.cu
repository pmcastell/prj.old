#include "hip/hip_runtime.h"
// MP 5 Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}
// Due Tuesday, January 22, 2013 at 11:59 p.m. PST

#include    <wb.h>

#define BLOCK_SIZE 1024 //@@ You can change this

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

__global__ void _scan(float *input, float *output, int len, float *blockSums, int numBlocks) {
	__shared__ float scan_array[2*BLOCK_SIZE];

	int stride, index,t=threadIdx.x,i=2*blockIdx.x*blockDim.x+t;
	scan_array[t]=(i<len ? input[i] : 0);
	scan_array[t+BLOCK_SIZE]=(i+BLOCK_SIZE<len ? input[i+BLOCK_SIZE] :0 );
	for(stride=1;stride<=BLOCK_SIZE;stride*=2) {
		__syncthreads();
		index=(t+1)*stride*2-1;
		if (index<2*BLOCK_SIZE) {
			scan_array[index]+=scan_array[index-stride];
		}
	}
	for(stride=BLOCK_SIZE/2;stride>0;stride/=2) {
		__syncthreads();
		index=(t+1)*stride*2-1;
		if (index+stride<2*BLOCK_SIZE) {
			scan_array[index+stride]+=scan_array[index];
		}
	}
	__syncthreads();
	if (blockSums!=0) {
		blockSums[blockIdx.x]=scan_array[2*BLOCK_SIZE-1];
	}
	if (i<len) {
		output[i]=scan_array[t];
	}
	if (i+BLOCK_SIZE<len) {
		output[i+BLOCK_SIZE]=scan_array[t+BLOCK_SIZE];
	}
}
__global__ void sum(float *input,float *blockSums,int len) {
	int i=2*blockIdx.x*blockDim.x+threadIdx.x;
	if (blockIdx.x>0 ) {
		if (i<len) {
			input[i]+=blockSums[blockIdx.x-1];
		}
		if (i+BLOCK_SIZE<len) {
			input[i+BLOCK_SIZE]+=blockSums[blockIdx.x-1];
		}
	}
}
void scan(float * input, float * output, int len) {
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here
	float *blockSumsInput, *blockSumsOutput;
	int numBlocks=((len-1)/(2*BLOCK_SIZE)+1);
    dim3 DimGrid(numBlocks, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
	hipMalloc((void**)&blockSumsInput,numBlocks*sizeof(float));
	hipMalloc((void**)&blockSumsOutput,numBlocks*sizeof(float));
	_scan<<<DimGrid,DimBlock>>>(input,output, len,blockSumsInput,numBlocks);
	dim3 DimGridB((numBlocks-1)/(2*BLOCK_SIZE)+1,1,1);
	dim3 DimBlockB(BLOCK_SIZE,1,1);
	_scan<<<DimGridB,DimBlockB>>>(blockSumsInput,blockSumsOutput,numBlocks,0,0);
	if (numBlocks>1) {
		sum<<<DimGrid,DimBlock>>>(output,blockSumsOutput,len);
	}

}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*) malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);
    wbLog(TRACE, "hostInput[0]: ", hostInput[0]);
    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements*sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements*sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements*sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements*sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
    scan(deviceInput,deviceOutput,numElements);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements*sizeof(float), hipMemcpyDeviceToHost));
    wbLog(TRACE, "hostOutput[0]: ", hostOutput[0]);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}

